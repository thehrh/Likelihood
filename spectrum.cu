#include "hip/hip_runtime.h"
/*
*Author: Maike Jung
*Date: 15.11.2016

*Purpose: create the arrival time spectrum of the neutrinos, that can then be used to
    generate random events: generateEvents.c
    calculate the likelihood for these events:  likelihood.c
    calculate the binned-likelihood:    binned_likelihood.c

SN - Model: Lawrence-Livermore
    time spectrum is convoluted with the first hit distribution, to account for not knowing the absolute arrival times

UNITS: mass: eV
       energy: MeV
       distance: Mpc
       time: s

add noise of 10-5!
*/

#include "spectrum.h"


/* time shift due to neutrino mass */
double getDeltaT(double E, double mass, double dist){
    double tDelta = dist*51.4635*(mass/E)*(mass/E);
    return tDelta;
}

double getTimeDelay(double t, double E, double mass, double dist){
    return t - getDeltaT(E, mass, dist);
}

double LL_time_spectrum_shifted(double t, double E, double mass, double dist){
    double time = getTimeDelay(t, E, mass, dist);
    if (time <= 0){
        // unphysical?
        return 0.0;
    }
    return LL_time_spectrum(time);
}

/* arrival time probability for a certain mass/distance - normalized */
double LLSpectrumTotal (double t, double E, double mass, double dist){
    return LL_time_spectrum_shifted(t, E, mass, dist)*LL_energy_spectrum(E);
}

void cumSumT(double *arrayToSum, double *cumulative){
    /*calculate the cumulative sum of the arrival time distribution*/
    int k, l;
    double cum;
    for (k = 0; k < REST; k++){
        cum = 0.0;
        for (l = 0; l <= k; l++){
            cum += arrayToSum[l];
        }
        cumulative[k] = cum;
    }
}

void firstHitDistWeightedArrivalTimeDist(double *arrivalTimeDist, double *cumulative, double events, double *result){
    int m;
    double count = 0.0;
    for (m = 0; m < REST; m++){
        result[m] = arrivalTimeDist[m]*events*pow((1 - cumulative[m]), events-1);
        count += result[m];
    }
    for (m = 0; m < REST; m++){
        result[m] = result[m]/count;
    }
}

/* calculate the probability to get the first hit after a certain amount of time */
void ProbFirstHitDist (double mass, double dist, double events, double *result){
    /*arrival time distribution of all the hits (for a certain mass) - project the E,t spectrum
    on the t axis - t in 0.01 steps from 0 to 10 seconds*/
    double totalArrivalTimeDist[REST];
    int i;
    double sum;
    double y, e;
    for (i = 0; i < REST; i++){
        /* set the sum to zero for each time bin */
        sum = 0.0;
        /*Integrate over the energy part for every time bin. We move in 0.01 MeV
        steps up to 60 MeV. For each pair of time and energy, we compute the
        product of time and energy PDF ("LLSpectrumTotal"), continually 
        incrementing the sum*/
        for (e = 0.01; e < EMAX; e += 0.01) {
            y = LLSpectrumTotal(i*STEPT, e, mass, dist);
            sum += y * 0.01;
        }
        totalArrivalTimeDist[i] = sum*STEPT;
    }

    double cumulative[REST];
    cumSumT(totalArrivalTimeDist, cumulative);

    firstHitDistWeightedArrivalTimeDist(totalArrivalTimeDist, cumulative, events, result);
}

void convolveHitDistWithLLTimeSpec(double *hitDist, double *convolSpec){
    int i, j;
    double pNew;
    /*perform the convolution*/
    for (i = 0; i < REST*1.3; i++){
        pNew = 0.0;
        for (j = 0; j < REST; j++){
            if ((i-0.3*REST + j) < REST && (i-0.3*REST + j) > 0){
                pNew += hitDist[j] * LL_time_spectrum( (j+i-0.3*REST)*STEPT );
            }
        convolSpec[i] = pNew;
        }
    }
}

/*calculate the correlation - new spectrum between -3 and 10s*/
/*this is stored in an array so newSpec[0] corresponds to a time of -3s
and newSpec[1.3*REST-1] to 10s*/
void correlation(double mass, double dist, double events, double *newSpec){
    double hitDist[REST];
    ProbFirstHitDist(mass, dist, events, hitDist);
    convolveHitDistWithLLTimeSpec(hitDist, newSpec);
}

void applyEnergyRes(int t, double *distribution, double *energySpectrum){
    int f, g;
    for (f=1; f<RESE; f+=1){
        double pNew = 0.0;
        for (g=-RESE; g<RESE+1; g+=5){
            if (f-g >= 0 && f-g <= RESE){
                pNew += GAUSS(g*STEPE, f*STEPE)*energySpectrum[f-g];
            }
            distribution[t*(RESE-1)+f-1] = pNew*STEPE;
        }
    }
}

/* This becomes the function we want to run in parallel on the GPU */
__global__
void getEnergySpec(double *mass, double *dist, double *timeArray, double *distribution, double *triggerEffs, bool *useEnergyRes){
	double time, pUnsmeared;
	int e, f, g, arrayIndex;

    int t = blockIdx.x;//*blockDim.x + threadIdx.x;
    if (t < REST ){
        double energySpectrum[RESE];
        energySpectrum[0] = 0.0;
        for (e=1; e<RESE; e++){
            // make this explicit for now, until know how to call function correctly
            time =  t*STEPT - (*dist)*51.4635*(*mass/e*STEPE)*(*mass/e*STEPE);//getTimeDelay(t*STEPT, e*STEPE, mass, dist);
            arrayIndex = (int) (time/(STEPT) + 0.3*REST);
            if (arrayIndex <= 0){
                arrayIndex = 0;
            }
            pUnsmeared = LL_energy_spectrum(e*STEPE)*timeArray[arrayIndex]*triggerEffs[e];
            if (!useEnergyRes){
                distribution[t*(RESE-1) +e-1] = pUnsmeared;
            }
            energySpectrum[e] = pUnsmeared;
        }
        if (useEnergyRes){
            // also make this explicit for now
            // applyEnergyRes(t, distribution, energySpectrum);
            for (f=1; f<RESE; f+=1){
                double pNew = 0.0;
                for (g=-RESE; g<RESE+1; g+=5){
                    if (f-g >= 0 && f-g <= RESE){
                        pNew += GAUSS(g*STEPE, f*STEPE)*energySpectrum[f-g];
                    }
                distribution[t*(RESE-1)+f-1] = pNew*STEPE;
                }
            }
        }
    }
}

void normalize(double *distribution){
	// normalize the spectrum to 1
	int k;
	double normalize = 0;

	for (k=0; k<(RESE-1)*REST; k++){
		normalize += distribution[k]*STEPT*STEPE;
	}

	for (k=0; k<(RESE-1)*REST; k++){
        distribution[k] *= 1.0/normalize;
    	}

}

/*generate the proper distribution*/
void generateDist(double mass, double dist, double events, double *distribution, double *triggerEffs, bool useEnergyRes){
    double *timeArray, *d_timeArray;
    timeArray = (double*)malloc((1.3*REST)*sizeof(double));
    double *d_mass, *d_dist, *d_distribution, *d_triggerEffs;
    bool *d_useEnergyRes;
    int size = sizeof(double);

    //double timeArray[(int) (1.3*REST)];
    correlation(mass, dist, events, timeArray);
    hipMalloc(&d_mass, size); hipMalloc(&d_dist, size);
    hipMalloc(&d_distribution, (RESE-1) * REST * size);
    hipMalloc(&d_triggerEffs, RESE*size);
    hipMalloc(&d_useEnergyRes, sizeof(bool));
    hipMalloc(&d_timeArray, (1.3*REST)*sizeof(double));

    hipMemcpy(d_mass, &mass, size, hipMemcpyHostToDevice);
    hipMemcpy(d_dist, &dist, size, hipMemcpyHostToDevice);
    hipMemcpy(d_timeArray, &timeArray, 1.3*REST*size, hipMemcpyHostToDevice);
    hipMemcpy(d_distribution, &distribution, (RESE-1) * REST * size, hipMemcpyHostToDevice);
    hipMemcpy(d_triggerEffs, &triggerEffs, RESE*size, hipMemcpyHostToDevice);
    hipMemcpy(d_useEnergyRes, &useEnergyRes, sizeof(bool), hipMemcpyHostToDevice);

    getEnergySpec<<< REST, 1 >>>(d_mass, d_dist, d_timeArray, d_distribution, d_triggerEffs, d_useEnergyRes);

    hipMemcpy(&distribution, d_distribution, (RESE-1) * REST * size, hipMemcpyDeviceToHost);

    hipFree(d_mass); hipFree(d_dist); hipFree(d_timeArray); hipFree(d_distribution);
    hipFree(d_triggerEffs); hipFree(d_useEnergyRes);

	normalize(distribution);
}
