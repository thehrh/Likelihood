#include "hip/hip_runtime.h"
/*
*Author: Maike Jung
*Date: 15.11.2016

*Purpose: create the arrival time spectrum of the neutrinos, that can then be used to
    generate random events: generateEvents.c
    calculate the likelihood for these events:  likelihood.c
    calculate the binned-likelihood:    binned_likelihood.c

SN - Model: Lawrence-Livermore
    time spectrum is convoluted with the first hit distribution, to account for not knowing the absolute arrival times

UNITS: mass: eV
       energy: MeV
       distance: Mpc
       time: s

add noise of 10-5!
*/

#include "spectrum.h"

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}


/* time shift due to neutrino mass */
user_data_t getDeltaT(user_data_t E, user_data_t mass, user_data_t dist){
    user_data_t tDelta = dist*51.4635*(mass/E)*(mass/E);
    return tDelta;
}

user_data_t getTimeDelay(user_data_t t, user_data_t E, user_data_t mass, user_data_t dist){
    return t - getDeltaT(E, mass, dist);
}

user_data_t LL_time_spectrum_shifted(user_data_t t, user_data_t E, user_data_t mass, user_data_t dist){
    user_data_t time = getTimeDelay(t, E, mass, dist);
    if (time <= 0){
        // unphysical?
        return 0.0;
    }
    return LL_time_spectrum(time);
}

/* arrival time probability for a certain mass/distance - normalized */
user_data_t LLSpectrumTotal (user_data_t t, user_data_t E, user_data_t mass, user_data_t dist){
    return LL_time_spectrum_shifted(t, E, mass, dist)*LL_energy_spectrum(E);
}

void cumSumT(user_data_t *arrayToSum, user_data_t *cumulative){
    /*calculate the cumulative sum of the arrival time distribution*/
    int k, l;
    user_data_t cum;
    for (k = 0; k < REST; k++){
        cum = 0.0;
        for (l = 0; l <= k; l++){
            cum += arrayToSum[l];
        }
        cumulative[k] = cum;
    }
}

void firstHitDistWeightedArrivalTimeDist(user_data_t *arrivalTimeDist, user_data_t *cumulative, user_data_t events, user_data_t *result){
    int m;
    user_data_t count = 0.0;
    for (m = 0; m < REST; m++){
        result[m] = arrivalTimeDist[m]*events*pow((1 - cumulative[m]), events-1);
        count += result[m];
    }
    for (m = 0; m < REST; m++){
        result[m] = result[m]/count;
    }
}

/* calculate the probability to get the first hit after a certain amount of time */
void ProbFirstHitDist (user_data_t mass, user_data_t dist, user_data_t events, user_data_t *result){
    /*arrival time distribution of all the hits (for a certain mass) - project the E,t spectrum
    on the t axis - t in 0.01 steps from 0 to 10 seconds*/
    user_data_t totalArrivalTimeDist[REST];
    int i;
    user_data_t sum;
    user_data_t y, e;
    for (i = 0; i < REST; i++){
        /* set the sum to zero for each time bin */
        sum = 0.0;
        /*Integrate over the energy part for every time bin. We move in 0.01 MeV
        steps up to 60 MeV. For each pair of time and energy, we compute the
        product of time and energy PDF ("LLSpectrumTotal"), continually 
        incrementing the sum*/
        for (e = 0.01; e < EMAX; e += 0.01) {
            y = LLSpectrumTotal(i*STEPT, e, mass, dist);
            sum += y * 0.01;
        }
        totalArrivalTimeDist[i] = sum*STEPT;
    }

    user_data_t cumulative[REST];
    cumSumT(totalArrivalTimeDist, cumulative);

    firstHitDistWeightedArrivalTimeDist(totalArrivalTimeDist, cumulative, events, result);
}

void convolveHitDistWithLLTimeSpec(user_data_t *hitDist, user_data_t *convolSpec){
    int i, j;
    user_data_t pNew;
    /*perform the convolution*/
    for (i = 0; i < REST*1.3; i++){
        pNew = 0.0;
        for (j = 0; j < REST; j++){
            if ((i-0.3*REST + j) < REST && (i-0.3*REST + j) > 0){
                pNew += hitDist[j] * LL_time_spectrum( (j+i-0.3*REST)*STEPT );
            }
        }
        convolSpec[i] = pNew;
    }
}

/*calculate the correlation - new spectrum between -3 and 10s*/
/*this is stored in an array so newSpec[0] corresponds to a time of -3s
and newSpec[1.3*REST-1] to 10s*/
void correlation(user_data_t mass, user_data_t dist, user_data_t events, user_data_t *newSpec){
    user_data_t hitDist[REST];
    ProbFirstHitDist(mass, dist, events, hitDist);
    convolveHitDistWithLLTimeSpec(hitDist, newSpec);
}

void applyEnergyRes(int t, user_data_t *distribution, user_data_t *energySpectrum){
    int f, g;
    for (f=1; f<RESE; f+=1){
        user_data_t pNew = 0.0;
        for (g=-RESE; g<RESE+1; g+=5){
            if (f-g >= 0 && f-g <= RESE){
                pNew += GAUSS(g*STEPE, f*STEPE)*energySpectrum[f-g];
            }
            distribution[t*(RESE-1)+f-1] = pNew*STEPE;
        }
    }
}


void normalize(user_data_t *distribution){
	// normalize the spectrum to 1
	int k;
	user_data_t normalize = 0;

	for (k=0; k<(RESE-1)*REST; k++){
		normalize += distribution[k]*STEPT*STEPE;
	}

	for (k=0; k<(RESE-1)*REST; k++){
		distribution[k] *= 1.0/normalize;
    	}
}


__global__
void getEnergySpec(user_data_t mass, user_data_t dist, user_data_t *timeArray, user_data_t *triggerEffs, user_data_t *distribution, bool useEnergyRes){
    user_data_t time, pUnsmeared, pNew, p_E_LL, p_t_LL, triggerEff;
    int e, f, g, arrayIndex;
    int t = blockIdx.x*blockDim.x + threadIdx.x;
    if (t < REST){
        user_data_t energySpectrum[RESE];
        energySpectrum[0] = 0.0;
        for (e=1; e<RESE; e++){
            // make this explicit for now, until know how to call function correctly
            time =  t*STEPT - dist*51.4635*(mass/(e*STEPE))*(mass/(e*STEPE));//getTimeDelay(t*STEPT, e*STEPE, mass, dist);
            arrayIndex = (int) (time/(STEPT) + 0.3*REST);
            if (arrayIndex <= 0){
                arrayIndex = 0;
	        }
            p_E_LL = LL_energy_spectrum(e*STEPE);
            p_t_LL = timeArray[arrayIndex];
            triggerEff = triggerEffs[e];
            /*
            if (t==0){
                printf("\n LL energy spectrum entry %d: %.10f \n", e, p_E_LL);
                printf("\n trigger Eff %d: %.10f \n", e, triggerEff);
                printf("\n timeArray entry %d: %.10f \n", arrayIndex, p_t_LL);
            }
            */
	        pUnsmeared = p_E_LL*p_t_LL*triggerEff;
            if (!useEnergyRes){
                distribution[t*(RESE-1) +e-1] = pUnsmeared;
            }
            energySpectrum[e] = pUnsmeared;
	    }
        if (useEnergyRes){
            // also make this explicit for now
            // applyEnergyRes(t, distribution, energySpectrum);
            for (f=1; f<RESE; f+=1){
                /*
                if (t==0){
                    printf("\n For t=0, energy spectrum entry %d: %.10f \n", f, energySpectrum[f]);
                }
                */
                pNew = 0.0;
                for (g=-RESE; g<RESE+1; g+=5){
                    if (f-g >= 0 && f-g < RESE){
                        pNew += GAUSS(g*STEPE, f*STEPE)*energySpectrum[f-g];
		            }
                }
                /*
                if (t==0){
                    printf("\n For t=0, write %.10f to %d \n", pNew*STEPE, t*(RESE-1)+f-1);
                }
                */
                distribution[t*(RESE-1)+f-1] = pNew*STEPE;
            }
	    }
        /*
        if (t==0){
            printf("\n spectrum[120]: %.10f", distribution[120]);
        }
        */
    }	
}

void generateDist(user_data_t mass, user_data_t dist, user_data_t events, user_data_t *distribution, user_data_t *triggerEffs, bool useEnergyRes){
    user_data_t timeArray[(int) (1.3*REST)];
    user_data_t *d_triggerEffs, *d_distribution, *d_timeArray;
    int size = sizeof(user_data_t);

    correlation(mass, dist, events, timeArray);

    /*
    //create a file from the timeArray for debugging
    char filename[sizeof "timeArray_CUDA.txt"];
    sprintf(filename, "timeArray_CUDA.txt");
    FILE *f = fopen(filename, "w+");
    for(int i=0; i<(int)(1.3*REST); i++){
        fprintf(f, "%e\n", timeArray[i]);
    }
    fclose(f);
    */

    hipMalloc((void **)&d_distribution, (RESE-1) * REST * size);
    hipMalloc((void **)&d_triggerEffs, (RESE+1)*size);
    hipMalloc((void **)&d_timeArray, 1.3*REST*size);

    hipMemcpy(d_timeArray, &timeArray, 1.3*REST*size, hipMemcpyHostToDevice);
    hipMemcpy(d_triggerEffs, triggerEffs, (RESE+1)*size, hipMemcpyHostToDevice);

    getEnergySpec<<<(REST + 511) / 512, 512>>>(mass, dist, d_timeArray, d_triggerEffs, d_distribution, useEnergyRes);
    CudaCheckError();

    hipMemcpy(distribution, d_distribution, (RESE-1) * REST * size, hipMemcpyDeviceToHost);

    /*
    //create a file from the dist before norm
    char filename2[sizeof "spec_before_norm_CUDA.txt"];
    sprintf(filename2, "spec_before_norm_CUDA.txt");
    FILE *f2 = fopen(filename2, "w+");
    for(int i=0; i<(RESE-1)*REST; i++){
        fprintf(f2, "%e\n", distribution[i]);
    }
    fclose(f2);
    */

    hipFree(d_timeArray);
    hipFree(d_distribution);
    hipFree(d_triggerEffs);

    normalize(distribution);
}
